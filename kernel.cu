#include "hip/hip_runtime.h"
#include "geometry.h"
#include "coating.h"


using namespace std;
using std::default_random_engine;
using std::normal_distribution;


#define CHECK_GPU(call) \
{ \
const hipError_t error = call; \
	if (error != hipSuccess) \
						{ \
		printf("\n\n");\
		printf("Error occured @ \n");\
		printf("FILE= %s   :   LINE= %d, \n", __FILE__, __LINE__); \
		printf("reason: %s\n\n", hipGetErrorString(error)); \
		printf("\n");\
		getchar();\
		exit(1);\
					}\
}\


#define CHECK_GPU_KERNEL(error) \
{ \
	if (error != hipSuccess) \
								{ \
		printf("\n\n");\
		printf("Error occured @ \n");\
		printf("FILE= %s   :   LINE= %d, \n", __FILE__, __LINE__); \
		printf("reason: %s\n\n",  hipGetErrorString(error)); \
		printf("\n");\
		getchar();\
		exit(1);\
						}\
}\

hipError_t error = hipSuccess;

struct Landing {
	double xl, yl, zl;
	unsigned int land_id = UINT_MAX;

	/*__host__ __device__*/  Landing() { land_id = UINT_MAX; }
	/*__host__ __device__*/ void reset() { land_id = UINT_MAX; }

};

struct particleprop {
	double rho_p;
	double cp_p;
	double mu_p; // viscosity of particle
	double k_p; // heat conduction coefficient of particle
	double sigma_p; // surface tension coefficient of particle
	double Hf_p; // latent heat of partcile
	double cangle_p; // liquid-solid contact angle


};


__global__ void deposit(coating::particle* p_arg, Triangle* tri_arg, Point* P_arg, int MAXNUM_arg, double Tsub_arg, particleprop parprop_arg, Landing* l_arg) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < MAXNUM_arg) {


		double nx = sin(p_arg->omega_p)*cos(p_arg->theta_p);
		double ny = sin(p_arg->omega_p)*sin(p_arg->theta_p);
		double nz = -cos(p_arg->omega_p);


		//if (tri_arg[tid].nx*nx + tri_arg[tid].ny*ny + tri_arg[tid].nz*nz < 0){
		//	continue; // the injected particle does not land on this plane
		//}
		// now find intersection of particle with the triangle plane

		// xp=x0+nx*t, yp=y0+ny*t, zp=z0+nz*t
		// a'x+b'y+c'z=a*'x'0+b'*y'0+c'*z'0

		double num = tri_arg[tid].nx*(tri_arg[tid].p0.x - p_arg->x_p) + tri_arg[tid].ny*(tri_arg[tid].p0.y - p_arg->y_p) + tri_arg[tid].nz*(tri_arg[tid].p0.z - p_arg->z_p);
		double denum = tri_arg[tid].nx*nx + tri_arg[tid].ny*ny + tri_arg[tid].nz*nz;

		double t = num / denum;

		double xpp = nx*t + p_arg->x_p;
		double ypp = ny*t + p_arg->y_p;
		double zpp = nz*t + p_arg->z_p;

		/*double x_p = coating::gun::z_g*tan(p_arg->omega_p)*cos(p_arg->theta_p) + coating::gun::x_g;
		double y_p = coating::gun::z_g*tan(p_arg->omega_p)*sin(p_arg->theta_p) + coating::gun::y_g;
		double z_p = -coating::gun::z_g;*/

		Point O(xpp, ypp, zpp);

		Triangle OP0P1(O, tri_arg[tid].p0, tri_arg[tid].p1);
		Triangle OP0P2(O, tri_arg[tid].p0, tri_arg[tid].p2);
		Triangle OP1P2(O, tri_arg[tid].p1, tri_arg[tid].p2);

		//cout << OP0P1.area() + OP0P2.area() + OP1P2.area() <<"   "<< tri_arg[tid].area() << endl;

		//if ((OP0P1.area() + OP0P2.area() + OP1P2.area() < tri_arg[tid].area() + 1e-6) && (OP0P1.area() + OP0P2.area() + OP1P2.area() > tri_arg[tid].area() - 1e-6)){
		
		if (tri_arg[tid].nx*nx + tri_arg[tid].ny*ny + tri_arg[tid].nz*nz < 0) {
			//if (abs((OP0P1.area() + OP0P2.area() + OP1P2.area()) - tri_arg[tid].area()) < 1e-5) {
			if (!( (OP0P1.area() + OP0P2.area() + OP1P2.area()) > tri_arg[tid].area() ) ) {
				l_arg->xl = xpp;
				l_arg->yl = ypp;
				l_arg->zl = zpp;
				l_arg->land_id = tid;

				//cout << "FOUND A MATCH!" << endl;
				//cout << xpp << "   " << ypp << "   " << zpp<<endl;


				p_arg->V_z = p_arg->V_p*cos(p_arg->omega_p);
				p_arg->V_x = p_arg->V_p*sin(p_arg->omega_p)*cos(p_arg->theta_p);
				p_arg->V_y = p_arg->V_p*sin(p_arg->omega_p)*sin(p_arg->theta_p);

				double We = parprop_arg.rho_p*p_arg->V_z*p_arg->V_z*p_arg->D_p / parprop_arg.sigma_p;
				double Re = parprop_arg.rho_p*p_arg->V_z*p_arg->D_p / parprop_arg.mu_p;
				double Pe = p_arg->V_z* p_arg->D_p *  parprop_arg.rho_p* parprop_arg.cp_p / parprop_arg.k_p;
				double St = parprop_arg.cp_p*(p_arg->T_p - Tsub_arg) / parprop_arg.Hf_p;

				double num = We + 12.0;
				double denum = 3.0*(1.0 - cos(parprop_arg.cangle_p)) + (4.0*We / sqrt(Re)) + We*sqrt(0.75*St / Pe);

				double spreadfactor = sqrt(num / denum);

				p_arg->dmax_p = spreadfactor*p_arg->D_p;
				p_arg->h_p = 2 * p_arg->D_p / (3.0*spreadfactor*spreadfactor);

				double PI = 3.14159265;

				tri_arg[tid].h = tri_arg[tid].h + (p_arg->h_p*(0.25*PI*p_arg->D_p*p_arg->D_p) / tri_arg[tid].area());

				P_arg[tri_arg[tid].pid[0]].h = tri_arg[tid].h;
				P_arg[tri_arg[tid].pid[1]].h = tri_arg[tid].h;
				P_arg[tri_arg[tid].pid[2]].h = tri_arg[tid].h;

				return;
			}

		}


	}

}



void deposit_cpu(coating::particle* p_arg, Triangle* tri_arg, int MAXNUM_arg, Landing* l_arg) {

	for (int tid = 0; tid < MAXNUM_arg; tid++) {

		double nx = sin(p_arg->omega_p)*cos(p_arg->theta_p);
		double ny = sin(p_arg->omega_p)*sin(p_arg->theta_p);
		double nz = -cos(p_arg->omega_p);


		//if (tri_arg[tid].nx*nx + tri_arg[tid].ny*ny + tri_arg[tid].nz*nz < 0){
		//	continue; // the injected particle does not land on this plane
		//}
		// now find intersection of particle with the triangle plane

		// xp=x0+nx*t, yp=y0+ny*t, zp=z0+nz*t
		// a'x+b'y+c'z=a*'x'0+b'*y'0+c'*z'0

		double num = tri_arg[tid].nx*(tri_arg[tid].p0.x - p_arg->x_p) + tri_arg[tid].ny*(tri_arg[tid].p0.y - p_arg->y_p) + tri_arg[tid].nz*(tri_arg[tid].p0.z - p_arg->z_p);
		double denum = tri_arg[tid].nx*nx + tri_arg[tid].ny*ny + tri_arg[tid].nz*nz;

		double t = num / denum;

		double xpp = nx*t + p_arg->x_p;
		double ypp = ny*t + p_arg->y_p;
		double zpp = nz*t + p_arg->z_p;

		double x_p = coating::gun::z_g*tan(p_arg->omega_p)*cos(p_arg->theta_p) + coating::gun::x_g;
		double y_p = coating::gun::z_g*tan(p_arg->omega_p)*sin(p_arg->theta_p) + coating::gun::y_g;
		double z_p = -coating::gun::z_g;

		Point O(xpp, ypp, zpp);

		Triangle OP0P1(O, tri_arg[tid].p0, tri_arg[tid].p1);
		Triangle OP0P2(O, tri_arg[tid].p0, tri_arg[tid].p2);
		Triangle OP1P2(O, tri_arg[tid].p1, tri_arg[tid].p2);

		//cout << OP0P1.area() + OP0P2.area() + OP1P2.area() <<"   "<< tri_arg[tid].area() << endl;

		//if ((OP0P1.area() + OP0P2.area() + OP1P2.area() < tri_arg[tid].area() + 1e-6) && (OP0P1.area() + OP0P2.area() + OP1P2.area() > tri_arg[tid].area() - 1e-6)){
		if (abs((OP0P1.area() + OP0P2.area() + OP1P2.area()) - tri_arg[tid].area())<1e-5) {
			l_arg->xl = xpp;
			l_arg->yl = ypp;
			l_arg->zl = zpp;
			l_arg->land_id = tid;

			p_arg->impact();
			p_arg->dmax_p = p_arg->spreadfactor()*p_arg->D_p;
			p_arg->h_p = 2 * p_arg->D_p / (3.0*p_arg->spreadfactor()*p_arg->spreadfactor());
			tri_arg[tid].h = tri_arg[tid].h + (p_arg->h_p*(0.25*PI*p_arg->D_p*p_arg->D_p) / tri_arg[tid].area());


			//cout << "FOUND A MATCH!" << endl;
			//cout << xpp << "   " << ypp << "   " << zpp<<endl;

			//return;
		}






	}

}



void write_file_triangles(Triangle* t_arg, Point* p_arg, unsigned int gi_arg) {
	ofstream f;
	stringstream filename;
	filename << "";
	filename << "output--" << gi_arg << ".dat";
	//filename << "output.dat";

	f.open(filename.str());

	f << "TITLE = \"Thickness plot\"" << endl;
	f << "VARIABLES = \"X\", \"Y\", \"Z\", \"h\" \"Z'\" " << endl;
	f << "ZONE N = " << NUM_POINTS << ", E = " << NUM_TRIANGLES << ", F = FEPOINT,  ET=TRIANGLE " << endl;

	/*for (unsigned int j = 0; j < NY; j++) {
	for (unsigned int i = 0; i < NX; i++) {

	unsigned int ij = i + NX*j;*/
	for (unsigned int i = 0; i < NUM_POINTS; i++) {

		f << p_arg[i].x << "   " << p_arg[i].y << "   " << p_arg[i].z << "   " << p_arg[i].h << "   " << p_arg[i].z+ p_arg[i].h << endl;
		//p_arg[t_arg[i].pid[0]].x  << "   " << p_arg[t_arg[i].pid[0]].y  << "   " << p_arg[t_arg[i].pid[0]].z  << "   " << t_arg[i].h << endl;
		//f << p_arg[t_arg[i].pid[1]].x  << "   " << p_arg[t_arg[i].pid[1]].y  << "   " << p_arg[t_arg[i].pid[1]].z << "   "  << t_arg[i].h << endl;
		//f << p_arg[t_arg[i].pid[2]].x  << "   " << p_arg[t_arg[i].pid[2]].y  << "   " << p_arg[t_arg[i].pid[2]].z << "   "  << t_arg[i].h << endl;
	}

	for (unsigned int i = 0; i < NUM_TRIANGLES; i++) {
		f << t_arg[i].pid[0] + 1 << "   " << t_arg[i].pid[1] + 1 << "   " << t_arg[i].pid[2] + 1 << endl;

	}

	f.flush();
	f.close();

}


using std::cout;
using namespace coating;



int main() {


	pt_data mydata;

	
	find_avglength("Cyl.stl");
	/*AVG_L = 0.0159931;
	NUM_TRIANGLES = 198899;
	NUM_POINTS_RAW = 596697;*/

	mydata = from_stl("Cyl.stl");
	Triangle* t = mydata.t;
	Point* P = mydata.p;

	find_neighbors(t, P, NUM_TRIANGLES);
	triangle_report(t, P, NUM_TRIANGLES);

	

	write_file_triangles(t, P, 50000);
	//cout << endl << "Press Enter ...";
	//getchar();

	//Triangle* t;
	//ifstream f("triangles.txt");
	//string s;
	//stringstream ss;


	//NUM_TRIANGLES = 198899;
	//f >> NUM_TRIANGLES;
	//f >> s;

	//getline(f, s);
	//ss << s;
	//ss >> NUM_TRIANGLES;
	//ss.clear();


	//t = new Triangle[NUM_TRIANGLES];
	//for (int i = 0; i < NUM_TRIANGLES; i++) {

	//	//f >> s;



	//	getline(f, s);

	//	ss << s;
	//	ss >> i >> t[i].nx >> t[i].ny >> t[i].nz >>
	//		t[i].p0.x >> t[i].p0.y >> t[i].p0.z >>
	//		t[i].p1.x >> t[i].p1.y >> t[i].p1.z >>
	//		t[i].p2.x >> t[i].p2.y >> t[i].p2.z >>
	//		t[i].neighbor[0] >> t[i].neighbor[1] >> t[i].neighbor[2];
	//	t[i].h = 0.0;
	//	ss.clear();
	//	//cout << "\r" << i;
	//}
	//f.close();
	//cout << endl;



	Triangle* t_dev;
	CHECK_GPU(hipMalloc((void **)&t_dev, NUM_TRIANGLES * sizeof(Triangle)));
	CHECK_GPU(hipMemcpy(t_dev, t, NUM_TRIANGLES * sizeof(Triangle), hipMemcpyHostToDevice));


	Point* P_dev;
	CHECK_GPU(hipMalloc((void **)&P_dev, NUM_POINTS * sizeof(Point)));
	CHECK_GPU(hipMemcpy(P_dev, P, NUM_POINTS * sizeof(Point), hipMemcpyHostToDevice));

	coating::particle* p_dev;
	CHECK_GPU(hipMalloc((void **)&p_dev, 1 * sizeof(coating::particle)));

	Landing l;
	l.xl = 0.0;
	l.yl = 0.0;
	l.zl = 0.0;


	Landing* l_dev;
	CHECK_GPU(hipMalloc((void **)&l_dev, 1 * sizeof(Landing)));
	CHECK_GPU(hipMemcpy(l_dev, &l, 1 * sizeof(Landing), hipMemcpyHostToDevice));



	// settings parameters
	coating::settings::Dmean = 50.0;
	coating::settings::Dstd = 10.0;

	coating::settings::Vmean = 60.0;
	coating::settings::Vstd = 5.7;

	coating::settings::Tmean = 1609.0;
	coating::settings::Tstd = 219.0;

	coating::settings::Omean = 0.0;
	coating::settings::Ostd = 5.0;



	// substrate parameters

	coating::Substrate::XMIN = 0.0;
	coating::Substrate::XMAX = 0.005;
	coating::Substrate::YMIN = 0.0;
	coating::Substrate::YMAX = 0.005;
	NX = 250;
	NY = 250;
	coating::Substrate::DELX = (coating::Substrate::XMAX - coating::Substrate::XMIN) / NX;
	coating::Substrate::DELY = (coating::Substrate::XMAX - coating::Substrate::XMIN) / NY;
	coating::Substrate::T_sub = 200.0;


	//coating::Substrate::h = new double[NX*NY];
	//h = new double[NX*NY];
	//
	//	for (unsigned int i = 0; i < NX*NY; i++) { /*coating::Substrate::h[i] = 0.0; */ h[i] = 0.0; }



	// gun parameters

	coating::gun::x_g = 0.0;
	coating::gun::y_g = 0.0;
	coating::gun::z_g = 10.0;

	coating::gun::u_g = 0.0;
	coating::gun::v_g = 0.0;
	coating::gun::w_g = 0.0;

	coating::gun::mfrate = 0.126*0.001;
	coating::gun::injtime = 0.005;


	// particle properties

	coating::particle::rho_p = 3000.0;;
	coating::particle::cp_p = 1300.0;;
	coating::particle::mu_p = 0.175E-4;; // viscosity of particle
	coating::particle::k_p = 60.0;; // heat conduction coefficient of particle
	coating::particle::sigma_p = 0.69;; // surface tension coefficient of particle
	coating::particle::Hf_p = 0.1075E7;; // latent heat of partcile
	coating::particle::cangle_p = 0.0;; // liquid-solid contact angle

	particleprop parprop;
	parprop.rho_p = particle::rho_p;
	parprop.cp_p = particle::cp_p;
	parprop.mu_p = particle::mu_p;
	parprop.k_p = particle::k_p;
	parprop.sigma_p = particle::sigma_p;
	parprop.Hf_p = particle::Hf_p;
	parprop.cangle_p = particle::cangle_p;


	// --------------------------------------------------------------------------------------------------------------- //



	unsigned int numtime = 1000;
	double dt = coating::gun::injtime / numtime;
	double time = 0.0;
	double mass_total = coating::gun::injtime*coating::gun::mfrate;
	coating::particle p;
	p.reset();
	CHECK_GPU(hipMemcpy(p_dev, &p, 1 * sizeof(coating::particle), hipMemcpyHostToDevice));


	//deposit_cpu(&p, t, NUM_TRIANGLES, &l);
	deposit << < int(NUM_TRIANGLES / 1024) + 1, 1024 >> > (p_dev, t_dev, P_dev, NUM_TRIANGLES, Substrate::T_sub, parprop, l_dev);
	error = hipGetLastError();
	CHECK_GPU_KERNEL(error);



	CHECK_GPU(hipMemcpy(&l, l_dev, 1 * sizeof(Landing), hipMemcpyDeviceToHost));


	//cout << t[l.land_id].area() << endl;


	double PI = 3.14159265;

	gi = 0;

	//while (time<coating::gun::injtime) {


	while (gi<numtime) {

		double mass_dt_init = coating::gun::mfrate*dt;
		double mass_dt = coating::gun::mfrate*dt;
		double mass_dt_old = mass_dt;

		gii = 0;
		tStart = cpuSecond();

		while (mass_dt >(PI / 6.0)*(1E-6)*(1E-6)*(1.E-6)) { // while remaining mass is larger than mass of 1 micron particle

			p.reset();
			CHECK_GPU(hipMemcpy(p_dev, &p, 1 * sizeof(coating::particle), hipMemcpyHostToDevice));


			//deposit_cpu(&p, t, NUM_TRIANGLES, &l);
			deposit << < int(NUM_TRIANGLES / 1024) + 1, 1024 >> > (p_dev, t_dev, P_dev, NUM_TRIANGLES, Substrate::T_sub, parprop, l_dev);
			error = hipGetLastError();
			CHECK_GPU_KERNEL(error);

			CHECK_GPU(hipMemcpy(&l, l_dev, 1 * sizeof(Landing), hipMemcpyDeviceToHost));


			mass_dt -= (PI / 6.0)*p.D_p*p.D_p*p.D_p;


			gii++;
			//cout << ii << "   " << mass_dt<< endl;

			//cout << p.D_p << "   " << p.T_p << "   " << p.theta_p << std::endl;

			//_getch();
			if (!(gii % 10000)) {
				//cout << "*-----------------------------------*" << endl;
				cout << "\rparticle#: " << gii << "   mass/mass0%= " << (mass_dt_old / mass_dt_init)*100.0;
				//cout << "*-----------------------------------*" << endl;
			}

			if (mass_dt> (PI / 6.0)*(1E-6)*(1E-6)*(1.E-6)) {
				mass_dt_old = mass_dt;
			}


		}


		cout << "\rparticle#: " << gii << "   mass/mass0%= " << (mass_dt_old / mass_dt_init)*100.0;
		gi++;
		cout << endl << "Ntime= " << gi << " time= " << time << "  / endtime= " << coating::gun::injtime << endl;
		tStop = cpuSecond();
		cout << "elapsed time: " << double(tStop - tStart) << " [milisec]" << endl;
		cout << "*-------------------------------------------*" << endl;


		//write_file(coating::Substrate::h, NX, NY);
		//write_file(h, NX, NY);
		if (!(gi % 100) || gi == 1) {
			//write_file(NX, NY);


			CHECK_GPU(hipMemcpy(t, t_dev, NUM_TRIANGLES * sizeof(Triangle), hipMemcpyDeviceToHost));
			CHECK_GPU(hipMemcpy(P, P_dev, NUM_POINTS * sizeof(Point), hipMemcpyDeviceToHost));
			write_file_triangles(t, P, gi);
		}

		//cout << "Press a key ..." << endl;
		//_getch();
		//Sleep(10 * 1000);

		time += dt;
	}

	//if (gi==numtime) write_file(NX, NY);
	//write_file(NX, NY);

	if (gi == numtime) {
		CHECK_GPU(hipMemcpy(t, t_dev, NUM_TRIANGLES * sizeof(Triangle), hipMemcpyDeviceToHost));
		CHECK_GPU(hipMemcpy(P, P_dev, NUM_POINTS * sizeof(Point), hipMemcpyDeviceToHost));
		write_file_triangles(t, P, gi);
	}
	CHECK_GPU(hipMemcpy(t, t_dev, NUM_TRIANGLES * sizeof(Triangle), hipMemcpyDeviceToHost));
	CHECK_GPU(hipMemcpy(P, P_dev, NUM_POINTS * sizeof(Point), hipMemcpyDeviceToHost));
	write_file_triangles(t, P, gi);

	//cout << gi << endl;
	//_getch();

	//delete[] coating::Substrate::h;
	delete[] h;


	hipFree(t_dev);
	hipFree(p_dev);
	hipFree(l_dev);


	_getch();

	return 0;
}






